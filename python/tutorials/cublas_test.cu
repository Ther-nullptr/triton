#include <stdio.h>
#include <sys/time.h>

#include <map>
#include <string>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>


template <typename T, typename S>
void allocate_memory(int b, int m, int n, int k, T **A, T **B, S **C) {
  hipMallocManaged(A, b * m * k * sizeof(T));
  hipMallocManaged(B, b * k * n * sizeof(T));
  hipMallocManaged(C, b * m * n * sizeof(S));
}

template <typename T, typename S> void free_memory(T *A, T *B, S *C) {
  hipFree(A);
  hipFree(B);
  hipFree(C);
}

template <typename T, typename S>
int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transA,
                          hipblasOperation_t transB, int b, int m, int n, int k,
                          T *A, T *B, S *C, int lda, int ldb, int ldc, S *alpha,
                          S *beta, int algo) {
  hipDataType AType, BType, CType, ComputeType;
  AType = BType = CType = ComputeType = HIP_R_16F;

  hipblasStatus_t status;
  status = hipblasGemmStridedBatchedEx(
      handle, transA, transB, m, n, k, alpha, A, AType, lda, m * k, B, BType,
      ldb, k * n, beta, C, CType, ldc, m * n, b, ComputeType,
      static_cast<hipblasGemmAlgo_t>(algo));

  if (status == HIPBLAS_STATUS_SUCCESS)
    return 1;
  else
    return -1;
}

template <typename T, typename S>
float test_gemm(hipblasHandle_t handle, int b, int m, int n, int k, T *A,
                       T *B, S *C, S *alpha, S *beta, int algo, int iteration) {
  float total_time = 0;
  for (int i = 0; i < iteration; ++i) {
    struct timeval start, end;
    hipDeviceSynchronize();
    hipProfilerStart();
    gettimeofday(&start, NULL);
    int success =
        cublas_gemm_ex(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, b, n, m, k, B, A, C, n,
                       k, n, alpha, beta, static_cast<hipblasGemmAlgo_t>(algo));
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    hipProfilerStop();
    if (success > 0 && i > 0) {
      total_time += (end.tv_sec - start.tv_sec) * 1000 +
                    (end.tv_usec - start.tv_usec) * 0.001;
    }
  }
//   if (total_time > 0) {
//     printf("algo %d: %.3f ms\n", algo, total_time / (iteration - 1));
//   }
  return total_time / (iteration - 1);
}

inline double tflops(int b, int m, int n, int k, float ms) {
  return 2 * (static_cast<double>(b) * m * n * k) * 1e-9 / static_cast<double>(ms); // / static_cast<double>(ms);
}

struct Shape {
  int b, m, n, k;
};

int main() {
  // initialize
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  setbuf(stdout, NULL);

  // test list of shape
  std::map<std::string, Shape> train_dict = {
      {"1-XxQKVw", {1, 8192, 4608, 12288}},
      {"2-QxK^T", {192, 512, 512, 128}},
      {"3-QK^TxV", {192, 512, 128, 512}},
      {"4-Proj", {1, 8192, 12288, 1536}},
      {"5-FC1", {1, 8192, 6144, 12288}},
      {"6-FC2", {1, 8192, 12288, 6144}},
      {"7-QxK^TFlat", {1, 512, 512, 24576}},
      {"8-QK^TxVFlat", {1, 512, 24576, 512}}};

  std::map<std::string, Shape> inference_w_o_KV_dict = {
      {"1-XxQKVw", {1, 8704, 4608, 12288}},
      {"2-QxK^T", {192, 543, 543, 128}},
      {"3-QK^TxV", {192, 543, 128, 543}},
      {"4-Proj", {1, 8688, 12288, 1536}},
      {"5-FC1", {1, 8688, 6144, 12288}},
      {"6-FC2", {1, 8688, 12288, 6144}},
      {"7-QxK^TFlat", {1, 543, 543, 24576}},
      {"8-QK^TxVFlat", {1, 543, 24576, 543}}};

  std::map<std::string, Shape> inference_w_KV_dict = {
      {"1-XxQKVw", {1, 16, 4608, 12288}},  {"2-QxK^T", {192, 1, 543, 128}},
      {"3-QK^TxV", {192, 1, 128, 543}},    {"4-Proj", {1, 16, 12288, 1536}},
      {"5-FC1", {1, 16, 6144, 12288}},     {"6-FC2", {1, 16, 12288, 6144}},
      {"7-QxK^TFlat", {1, 1, 543, 24576}}, {"8-QK^TxVFlat", {1, 1, 24576, 543}}};

  for (const auto &dict :
       {train_dict, inference_w_o_KV_dict, inference_w_KV_dict}) {
    printf("=========================\n");
    for (auto it = dict.begin(); it != dict.end(); ++it) {
      auto item = *it;
      printf("-------------------------\n");
      printf("%s: [%d, %d, %d, %d]\n", item.first.c_str(), item.second.b,
             item.second.m, item.second.n, item.second.k);

      Shape shape = item.second;
      int b = shape.b;
      int m = shape.m;
      int n = shape.n;
      int k = shape.k;

      int start_algo = HIPBLAS_GEMM_DEFAULT;
      int end_algo = CUBLAS_GEMM_ALGO23;
      int start_algo_t_op = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
      int end_algo_t_op = CUBLAS_GEMM_ALGO15_TENSOR_OP;
      int iteration = 100;

      half *hA, *hB, *hC;
      half h_alpha = __float2half_rn(1.0), h_beta = __float2half_rn(0.0);
      allocate_memory(b, m, n, k, &hA, &hB, &hC);
      for (int i = 0; i < m * k; ++i) {
        hA[i] = __float2half_rn(float(i % 255 - 127) / 127);
      }
      for (int i = 0; i < k * n; ++i) {
        hB[i] = __float2half_rn(float(i % 255 - 127) / 127);
      }

      printf(">>>>>>>>>>>>>>>>> test fp16 >>>>>>>>>>>>>>>>>\n");
      float min_time = 0xffff;
      hipblasGemmAlgo_t algo_index;
      for (int algo = start_algo; algo <= end_algo; ++algo) {
        float current_time = test_gemm(handle, b, m, n, k, hA, hB, hC, &h_alpha,
                                       &h_beta, algo, iteration);
        if (current_time < min_time) {
          min_time = current_time;
          algo_index = static_cast<hipblasGemmAlgo_t>(algo);
        }
      }
      for (int algo = start_algo_t_op; algo <= end_algo_t_op; ++algo) {
        float current_time = test_gemm(handle, b, m, n, k, hA, hB, hC, &h_alpha,
                                       &h_beta, algo, iteration);
        if (current_time < min_time) {
          min_time = current_time;
          algo_index = static_cast<hipblasGemmAlgo_t>(algo);
        }
      }
      printf("[%s] min_time: %.3f ms, best algorithm: %d, tflops: %.3lf\n", item.first.c_str(),
             min_time, static_cast<int>(algo_index), tflops(b, m, n, k, min_time));
      free_memory(hA, hB, hC);
    }
  }

  return 0;
}