#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

int main() {
  int iteration = 10;

  float min_time = 0xffff;
  hipblasGemmAlgo_t algo_index;
  for (const auto &algo : algoList) {
    float total_time = 0.0;
    for (int i = 0; i < iteration; i++) {

      hipEvent_t start, end;
      hipEventCreate(&start);
      hipEventCreate(&end);

      hipEventRecord(start, 0);
      hipblasGemmStridedBatchedEx(
          handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_a, HIP_R_16F, k,
          m * k, d_b, HIP_R_16F, n, k * n, &beta, d_c, HIP_R_16F, n, m * n,
          batch_count, HIP_R_16F, static_cast<hipblasGemmAlgo_t>(algo));
      hipEventRecord(end, 0);
      hipEventSynchronize(end);
      float elapsed_time;
      hipEventElapsedTime(&elapsed_time, start, end);
      total_time += elapsed_time;
    }
    float current_time = total_time / iteration;
    std::cout << "algo:" << algo << " " << current_time << " ms" << std::endl;
    if (current_time < min_time) {
      min_time = current_time;
      algo_index = algo;
    }
  }
  std::cout << "best:" << algo_index << " " << min_time << " ms" << std::endl;
}