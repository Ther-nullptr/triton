#include <stdio.h>
#include <sys/time.h>

#include <string>
#include <map>

#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

 
template <typename T, typename S>
void allocate_memory(int b, int m, int n, int k, T **A, T **B, S **C) {
    hipMallocManaged(A, b * m * k * sizeof(T));
    hipMallocManaged(B, b * k * n * sizeof(T));
    hipMallocManaged(C, b * m * n * sizeof(S));
}
 
template <typename T, typename S>
void free_memory(T *A, T *B, S *C) {
    hipFree(A);
    hipFree(B);
    hipFree(C);
}
 
template <typename T, typename S>
inline int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transA, hipblasOperation_t transB,
                   int b, int m, int n, int k, T *A, T *B, S *C, int lda, int ldb, int ldc,
                   S *alpha, S *beta, int algo) {
    hipDataType AType, BType, CType, ComputeType;
    AType = BType = CType = ComputeType = HIP_R_16F;

    hipblasStatus_t status;
    status = hipblasGemmStridedBatchedEx(handle,
                          transA,
                          transB,
                          m,
                          n,
                          k,
                          alpha,
                          A,
                          AType,
                          lda,
                          m * k,
                          B,
                          BType,
                          ldb,
                          k * n,
                          beta,
                          C,
                          CType,
                          ldc,
                          m * n,
                          b,
                          ComputeType,
                          static_cast<hipblasGemmAlgo_t>(algo));
    
    if (status == HIPBLAS_STATUS_SUCCESS)
        return 1;
    else
        return -1;
}
 
template <typename T, typename S>
inline float test_gemm(hipblasHandle_t handle, int b, int m, int n, int k, T *A, T *B, S *C,
               S *alpha, S *beta, int algo, int iteration) {
    float total_time = 0;
    for (int i = 0; i < iteration; ++i) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);
        int success = cublas_gemm_ex(handle,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     b, 
                                     n,
                                     m,
                                     k,
                                     B,
                                     A,
                                     C,
                                     n,
                                     k,
                                     n,
                                     alpha,
                                     beta,
                                     static_cast<hipblasGemmAlgo_t>(algo));
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, end);
        if (success > 0 && i > 0)
        {
            total_time += elapsed_time;
        }
    }
    if (total_time > 0)
    {
        printf("algo %d: %.3f ms\n", algo, total_time / (iteration - 1));
    }
    return total_time / (iteration - 1);
}

struct Shape
{
    int b, m, n, k;
};
 
int main() 
{
    // initialize
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // test list of shape
    std::map<std::string, Shape> train_dict = {
        {"XxQKVw", {1, 8192, 4608, 12288}},
        {"QxK^T", {192, 512, 512, 128}},
        {"QK^TxV", {192, 512, 128, 512}},
        {"Proj", {1, 8192, 12288, 1536}},
        {"FC1", {1, 8192, 6144, 12288}},
        {"FC2", {1, 8192, 12288, 6144}},
        {"QxK^TFlat", {1, 512, 512, 24576}},
        {"QK^TxVFlat", {1, 512, 24576, 512}}
    };

    std::map<std::string, Shape> inference_w_o_KV_dict = {
        {"XxQKVw", {1, 8704, 4608, 12288}},
        {"QxK^T", {192, 543, 543, 128}},
        {"QK^TxV", {192, 543, 128, 543}},
        {"Proj", {1, 8688, 12288, 1536}},
        {"FC1", {1, 8688, 6144, 12288}},
        {"FC2", {1, 8688, 12288, 6144}},
        {"QxK^TFlat", {1, 543, 543, 24576}},
        {"QK^TxVFlat", {1, 543, 24576, 543}}
    };

    std::map<std::string, Shape> inference_w_KV_dict = {
        {"XxQKVw", {1, 16, 4608, 12288}},
        {"QxK^T", {192, 1, 543, 128}},
        {"QK^TxV", {192, 1, 128, 543}},
        {"Proj", {1, 16, 12288, 1536}},
        {"FC1", {1, 16, 6144, 12288}},
        {"FC2", {1, 16, 12288, 6144}},
        {"QxK^TFlat", {1, 1, 543, 24576}},
        {"QK^TxVFlat", {1, 1, 24576, 543}}
    };
    
    for (const auto &dict : {train_dict, inference_w_o_KV_dict, inference_w_KV_dict})
    {
        printf("=========================\n");
        for (const auto &item : dict)
        {
            printf("-------------------------\n");
            printf("%s: [%d, %d, %d, %d]\n", item.first.c_str(), item.second.b, item.second.m, item.second.n, item.second.k);

            Shape shape = item.second;
            int b = shape.b;
            int m = shape.m;
            int n = shape.n;
            int k = shape.k;

            int start_algo = HIPBLAS_GEMM_DEFAULT;
            int end_algo = CUBLAS_GEMM_ALGO23;
            int start_algo_t_op = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
            int end_algo_t_op = CUBLAS_GEMM_ALGO15_TENSOR_OP;
            int iteration = 100;
        
            half *hA, *hB, *hC;
            half h_alpha = __float2half_rn(1.0), h_beta = __float2half_rn(0.0);
            allocate_memory(b, m, n, k, &hA, &hB, &hC);
            for (int i = 0; i < m * k; ++i) {
                hA[i] = __float2half_rn(float(i % 255 - 127) / 127);
            } 
            for (int i = 0; i < k * n; ++i) {
                hB[i] = __float2half_rn(float(i % 255 - 127) / 127);
            }
            

            // warm up
            printf(">>>>>>>>>>>>>>>>> warm up >>>>>>>>>>>>>>>>>\n");
            for (int algo = start_algo; algo <= end_algo; ++algo)
                test_gemm(handle, b, m, n, k, hA, hB, hC, &h_alpha, &h_beta, algo, 1);
            
            printf(">>>>>>>>>>>>>>>>> test fp16 >>>>>>>>>>>>>>>>>\n");
            float min_time = 0xffff;
            hipblasGemmAlgo_t algo_index;
            for (int algo = start_algo; algo <= end_algo; ++algo)
            {
                float current_time = test_gemm(handle, b, m, n, k, hA, hB, hC, &h_alpha, &h_beta, algo, iteration);
                if (current_time < min_time)
                {
                    min_time = current_time;
                    algo_index = static_cast<hipblasGemmAlgo_t>(algo);
                }
            }
            for (int algo = start_algo_t_op; algo <= end_algo_t_op; ++algo)
            {
                float current_time = test_gemm(handle, b, m, n, k, hA, hB, hC, &h_alpha, &h_beta, algo, iteration);
                if (current_time < min_time)
                {
                    min_time = current_time;
                    algo_index = static_cast<hipblasGemmAlgo_t>(algo);
                }
            }
            printf("[%s] min_time: %.3f ms, best algorithm: %d\n", item.first.c_str(), min_time, static_cast<int>(algo_index));
            free_memory(hA, hB, hC);
        }
    }
    
    return 0;
}